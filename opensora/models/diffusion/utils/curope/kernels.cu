#include "hip/hip_runtime.h"
/* 
  Copyright (C) 2022-present Naver Corporation. All rights reserved.
  Licensed under CC BY-NC-SA 4.0 (non-commercial use only).
*/

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(tensor) {\
    TORCH_CHECK((tensor).is_cuda(), #tensor " is not in cuda memory"); \
    TORCH_CHECK((tensor).is_contiguous(), #tensor " is not contiguous"); }
void CHECK_KERNEL() {auto error = hipGetLastError(); TORCH_CHECK( error == hipSuccess, hipGetErrorString(error));}


template < typename scalar_t  >
__global__ void rope_2d_cuda_kernel( 
        //scalar_t* __restrict__ tokens, 
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> tokens,
        const int64_t* __restrict__ pos, 
        const float base, 
        const float fwd )
        // const int N, const int H, const int D )
{
    // tokens shape = (B, N, H, D)
    const int N = tokens.size(1);
    const int H = tokens.size(2);
    const int D = tokens.size(3);
    
    // each block update a single token, for all heads
    // each thread takes care of a single output
    extern __shared__ float shared[];
    float* shared_inv_freq = shared + D;

    const int b = blockIdx.x / N;
    const int n = blockIdx.x % N;

    const int Q = D / 4; 
    // one token = [0..Q : Q..2Q : 2Q..3Q : 3Q..D]
    //              u_Y     v_Y     u_X      v_X

    // shared memory: first, compute inv_freq
    if (threadIdx.x < Q)
        shared_inv_freq[threadIdx.x] = fwd / powf(base, threadIdx.x/float(Q));
    __syncthreads();

    // start of X or Y part
    const int X = threadIdx.x < D/2 ? 0 : 1; 
    const int m = (X*D/2) + (threadIdx.x % Q);   // index of u_Y or u_X

    // grab the cos,sin appropriate for me
    const float freq = pos[blockIdx.x*2+X] * shared_inv_freq[threadIdx.x % Q];
    const float cos = cosf(freq);
    const float sin = sinf(freq);
    /*
    float* shared_cos_sin = shared + D + D/4;
    if ((threadIdx.x % (D/2)) < Q)
        shared_cos_sin[m+0] = cosf(freq);
    else
        shared_cos_sin[m+Q] = sinf(freq);
    __syncthreads();
    const float cos = shared_cos_sin[m+0];
    const float sin = shared_cos_sin[m+Q];
    */

    for (int h = 0; h < H; h++)
    {
        // then, load all the token for this head in shared memory
        shared[threadIdx.x] = tokens[b][n][h][threadIdx.x];
        __syncthreads();

        const float u = shared[m];
        const float v = shared[m+Q];
        
        // write output
        if ((threadIdx.x % (D/2)) < Q)
            tokens[b][n][h][threadIdx.x] = u*cos - v*sin;
        else
            tokens[b][n][h][threadIdx.x] = v*cos + u*sin;
    }
}

void rope_2d_cuda( torch::Tensor tokens, const torch::Tensor pos, const float base, const float fwd ) 
{
    const int B = tokens.size(0); // batch size
    const int N = tokens.size(1); // sequence length
    const int H = tokens.size(2); // number of heads
    const int D = tokens.size(3); // dimension per head

    TORCH_CHECK(tokens.stride(3) == 1 && tokens.stride(2) == D, "tokens are not contiguous");
    TORCH_CHECK(pos.is_contiguous(), "positions are not contiguous");
    TORCH_CHECK(pos.size(0) == B && pos.size(1) == N && pos.size(2) == 2, "bad pos.shape");
    TORCH_CHECK(D % 4 == 0, "token dim must be multiple of 4");

    // one block for each layer, one thread per local-max
    const int THREADS_PER_BLOCK = D;
    const int N_BLOCKS = B * N; // each block takes care of H*D values
    const int SHARED_MEM = sizeof(float) * (D + D/4);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(tokens.type(), "rope_2d_cuda", ([&] {
        rope_2d_cuda_kernel<scalar_t> <<<N_BLOCKS, THREADS_PER_BLOCK, SHARED_MEM>>> (
            //tokens.data_ptr<scalar_t>(), 
            tokens.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            pos.data_ptr<int64_t>(), 
            base, fwd); //, N, H, D );
    }));
}
